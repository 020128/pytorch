#include "hip/hip_runtime.h"
#include <algorithm>

#include <cub/block/block_reduce.cuh>
#include <cub/device/device_reduce.cuh>
#include <cub/device/device_scan.cuh>

#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/sgd/adagrad_op.h"
#include "caffe2/utils/math.h"

#ifdef __HIP_PLATFORM_HCC__
#define SEGREDUCE_MINBLOCKS 8
#else
#define SEGREDUCE_MINBLOCKS 16
#endif

namespace caffe2 {

namespace {

void inclusive_scan_wrapper(
    const int* length_data,
    int len_length,
    Tensor* temp_buffer,
    Tensor* prefix_sum_out,
    HIPContext* context_) {
  // Retrieve buffer size
  size_t temp_storage_bytes = 0;
  hipcub::DeviceScan::InclusiveSum(
      NULL,
      temp_storage_bytes,
      length_data,
      prefix_sum_out->template mutable_data<int>(),
      len_length,
      context_->hip_stream());
  // Allocate temporary storage
  auto buffer_size = (temp_storage_bytes + sizeof(int)) / sizeof(int);
  temp_buffer->Resize(buffer_size);
  void* d_temp_storage =
      static_cast<void*>(temp_buffer->template mutable_data<int>());
  // Run inclusive prefix sum
  hipcub::DeviceScan::InclusiveSum(
      d_temp_storage,
      temp_storage_bytes,
      length_data,
      prefix_sum_out->template mutable_data<int>(),
      len_length,
      context_->hip_stream());
}

template <typename SIndex, typename THalf, typename T, bool ExactBlock = false>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_2(1024, SEGREDUCE_MINBLOCKS)
#endif
__global__ void sparse_adagrad_fused_length_sum_gradient_kernel(
    const int* __restrict__ prefix_sum_length_data, // prefix of lengths
                                                    // (offsets for the
                                                    // segments)
    int N, // number of rows (hash size) of embedding table
    int post, // embedding dimension size
    int len_length, // number of segments
    const float epsilon,
    THalf* param,
    THalf* param_mom,
    const SIndex* indices,
    const T* __restrict__ grad,
    const float* lr) {
  const float LR = lr[0];
  // len_length blocks, each block process one segment
  int group = blockIdx.x; // the group-th segment
  int start = group == 0
      ? 0
      : prefix_sum_length_data[group - 1]; // start offset of the segment
  int end = prefix_sum_length_data[group]; // end offset of the segment
  CUDA_KERNEL_ASSERT(start <= N);
  CUDA_KERNEL_ASSERT(end <= N);

  if (ExactBlock) {
    const size_t gradIdx = group * post + threadIdx.x; // index for grad
    for (int line = start + threadIdx.y; line < end; line += blockDim.y) {
      // line: the idx in the indices
      // i: index in the embedding dimension, which is equal to threadIdx.x
      const SIndex index =
          indices[line]; // the index-th row in the embedding table
      const size_t paramIdx = index * post + threadIdx.x; // index for param

      float mom_new = grad[gradIdx] * grad[gradIdx] + param_mom[paramIdx];
      param_mom[paramIdx] = mom_new;
      float param_new =
          LR * grad[gradIdx] / (sqrtf(mom_new) + epsilon) + param[paramIdx];
      param[paramIdx] = param_new;
    }
  } else {
    for (int i = threadIdx.x; i < post; i += blockDim.x) {
      // i: index in the embedding dimension
      const size_t gradIdx = group * post + i; // index for grad
      for (int line = start; line < end; ++line) {
        // line: the idx in the indices
        const SIndex index =
            indices[line]; // the index row in the embedding table
        const size_t paramIdx = index * post + i; // index for param

        float mom_new = grad[gradIdx] * grad[gradIdx] + param_mom[paramIdx];
        param_mom[paramIdx] = mom_new;
        float param_new =
            LR * grad[gradIdx] / (sqrtf(mom_new) + epsilon) + param[paramIdx];
        param[paramIdx] = param_new;
      }
    }
  }
}

template <typename SIndex, typename THalf, typename T, int NumThreads>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_2(1024, SEGREDUCE_MINBLOCKS)
#endif
__global__ void sparse_adagrad_fused_length_weighted_sum_gradient_kernel(
    const int* __restrict__ prefix_sum_length_data,
    int N, // number of rows (hash size) of embedding table
    int post, // embedding dimension size
    int len_length, // number of segments
    const float epsilon,
    THalf* param,
    THalf* param_mom,
    const SIndex* indices,
    const T* __restrict__ grad,
    const T* __restrict__ weights,
    T* __restrict__ weights_grad_out,
    const float* lr) {
  const float LR = lr[0];
  // len_length blocks, each block process one segment
  int group = blockIdx.x; // the group-th segment
  int start = group == 0
      ? 0
      : prefix_sum_length_data[group - 1]; // start offset of the segment
  int end = prefix_sum_length_data[group]; // end offset of the segment
  CUDA_KERNEL_ASSERT(start <= N);
  CUDA_KERNEL_ASSERT(end <= N);

  // TODO: Tuning NumThreads for w_grad
  typedef hipcub::BlockReduce<float, NumThreads> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  // TODO(jianyuhuang): parallelize this outer loop
  for (int line = start; line < end; ++line) {
    T w_grad = 0;
    // line: the idx in the indices
    const SIndex index =
        indices[line]; // the index-th row in the embedding table

    // SparseAdagradFusedWithSparseLengthsWeightedSumGradientOp also fuses
    // LengthsRangeFill + Gather operator. In the normal SLWS operator weight is
    // accessed via weights[line] but in most cases the weights are
    // generated by LengthsRangeFill and Gather operator.
    // For example, if lengths is [2, 3, 1] LengthsRangeFill will generate [0,
    // 1; 0, 1, 2; 0] and they are used as indices of Gather.
    // So if we fuse all of these, weights[line] just becomes
    // weights[line - start].
    auto in_weight_temp = weights[line - start];

    for (int i = threadIdx.x; i < post; i += blockDim.x) {
      // i: index in the embedding dimension
      const size_t gradIdx = group * post + i; // index for in_grad
      const size_t paramIdx = index * post + i; // index for param

      // TODO: trying to reduce the variable number (common subexpression
      // elimination).
      auto in_grad_temp = grad[gradIdx];
      auto out_grad_temp = in_weight_temp * in_grad_temp;
      w_grad += in_grad_temp * param[paramIdx];

      // TODO: split it into two kernels to make it more similar to exact fusion
      // kernel (not Approx on CPUs).
      float mom_new = out_grad_temp * out_grad_temp + param_mom[paramIdx];
      param_mom[paramIdx] = mom_new;
      float param_new =
          LR * out_grad_temp / (sqrtf(mom_new) + epsilon) + param[paramIdx];
      param[paramIdx] = param_new;
    }

    w_grad = BlockReduce(temp_storage).Reduce(w_grad, hipcub::Sum());

    // Alternative:
    // int valid = min(post, blockDim.x);
    // float w_reduce_result = BlockReduce(temp_storage).Sum(w_grad, valid);

    if (threadIdx.x == 0) {
      weights_grad_out[line] = w_grad;
    }
    __syncthreads();
  }
}

template <typename SIndex, typename THalf, typename T>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_2(1024, SEGREDUCE_MINBLOCKS)
#endif
__global__ void rowwise_sparse_adagrad_fused_length_sum_gradient_kernel(
    const int* __restrict__ prefix_sum_length_data, // prefix of lengths
                                                    // (offsets for the
                                                    // segments)
    int N, // number of rows (hash size) of embedding table
    int post, // embedding dimension size
    int len_length, // number of segments
    const float epsilon,
    THalf* param,
    THalf* param_mom,
    const SIndex* indices,
    const T* __restrict__ grad,
    const float* lr) {
  const float LR = lr[0];
  // len_length blocks, each block process one segment
  int group = blockIdx.x; // the group-th segment
  int start = group == 0
      ? 0
      : prefix_sum_length_data[group - 1]; // start offset of the segment
  int end = prefix_sum_length_data[group]; // end offset of the segment
  CUDA_KERNEL_ASSERT(start <= N);
  CUDA_KERNEL_ASSERT(end <= N);

  // TODO: Tuning NumThreads for sum_squares
  typedef hipcub::BlockReduce<float, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ BlockReduce::TempStorage temp_storage;
  // int valid = min(post, CAFFE_CUDA_NUM_THREADS);
  int valid = min(post, blockDim.x);

  for (int line = start; line < end; ++line) {
    // line: the idx in the indices
    const SIndex index = indices[line]; // the index row in the embedding table
    float sum_squares = 0.0;
    __shared__ float row_sum_squares_avg;

    for (int i = threadIdx.x; i < post; i += blockDim.x) {
      // i: index in the embedding dimension
      const float x_ij = grad[group * post + i];
      sum_squares += x_ij * x_ij;
    }
    float reduce_result = BlockReduce(temp_storage).Sum(sum_squares, valid);
    // float reduce_result =
    //     BlockReduce(temp_storage).Sum(sum_squares, blockDim.x);

    if (threadIdx.x == 0) {
      row_sum_squares_avg = reduce_result / static_cast<float>(post);
      param_mom[index] += row_sum_squares_avg;
    }
    __syncthreads();

    // update param
    float step = LR / (sqrtf(param_mom[index]) + epsilon);
    for (int i = threadIdx.x; i < post; i += blockDim.x) {
      const size_t paramIdx = index * post + i; // index for param
      param[paramIdx] = param[paramIdx] + grad[group * post + i] * step;
    }
  }
}

template <typename SIndex, typename THalf, typename T, int NumThreads>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_2(1024, SEGREDUCE_MINBLOCKS)
#endif
__global__
    void rowwise_sparse_adagrad_fused_length_weighted_sum_gradient_kernel(
        const int* __restrict__ prefix_sum_length_data, // prefix of lengths
                                                        // (offsets for the
                                                        // segments)
        int N, // number of rows (hash size) of embedding table
        int post, // embedding dimension size
        int len_length, // number of segments
        const float epsilon,
        THalf* param,
        THalf* param_mom,
        const SIndex* indices,
        const T* __restrict__ grad,
        const T* __restrict__ weights,
        T* __restrict__ weights_grad_out,
        const float* lr) {
  const float LR = lr[0];
  // len_length blocks, each block process one segment
  int group = blockIdx.x; // the group-th segment
  int start = group == 0
      ? 0
      : prefix_sum_length_data[group - 1]; // start offset of the segment
  int end = prefix_sum_length_data[group]; // end offset of the segment
  CUDA_KERNEL_ASSERT(start <= N);
  CUDA_KERNEL_ASSERT(end <= N);

  // TODO: Tuning NumThreads for w_grad
  typedef hipcub::BlockReduce<float, NumThreads> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  // for avg_square_weight. Can we reuse temp_storage
  __shared__ typename BlockReduce::TempStorage temp_storage2;
  // Why do we need to add typename here?

  // TODO(jianyuhuang): parallelize this outer loop
  for (int line = start; line < end; ++line) {
    T w_grad = 0;
    // i: index in the embedding dimension
    const SIndex index = indices[line];

    auto in_weight_temp = weights[line - start];

    float sum_squares = 0.0;
    __shared__ float row_sum_squares_avg;

    for (int i = threadIdx.x; i < post; i += blockDim.x) {
      const float x_ij = grad[group * post + i];
      sum_squares += x_ij * x_ij;
    }
    // float reduce_result = BlockReduce(temp_storage).Sum(sum_squares, valid);
    float reduce_result =
        BlockReduce(temp_storage2).Sum(sum_squares, NumThreads);

    if (threadIdx.x == 0) {
      row_sum_squares_avg = reduce_result / static_cast<float>(post);
      param_mom[index] += row_sum_squares_avg * in_weight_temp * in_weight_temp;
    }
    __syncthreads();

    // update param
    float step = LR / (sqrtf(param_mom[index]) + epsilon);

    for (int i = threadIdx.x; i < post; i += blockDim.x) {
      const size_t gradIdx = group * post + i; // index for in_grad
      const size_t paramIdx = index * post + i; // index for param
      // TODO: trying to reduce the variable number (common subexpression
      // elimination).
      auto in_grad_temp = grad[gradIdx];
      auto out_grad_temp = in_weight_temp * in_grad_temp;
      w_grad += in_grad_temp * param[paramIdx];

      // TODO: split it into two kernels to make it more similar to exact fusion
      // kernel (not Approx on CPUs).
      param[paramIdx] = out_grad_temp * step + param[paramIdx];
    }
    w_grad = BlockReduce(temp_storage).Reduce(w_grad, hipcub::Sum());

    // int valid = min(post, blockDim.x);
    // float w_reduce_result = BlockReduce(temp_storage).Sum(w_grad, valid);

    if (threadIdx.x == 0) {
      weights_grad_out[line] = w_grad;
    }
    __syncthreads();
  }
}

} // namespace

__global__ void AdagradUpdate(
    int N,
    const float* w,
    const float* g,
    const float* h,
    float* nw,
    float* nh,
    float epsilon,
    float decay,
    const float* lr) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float hi = nh[i] = decay * h[i] + gi * gi;
    nw[i] = w[i] + lr[0] * gi / (sqrtf(hi) + epsilon);
  }
}

template <>
void adagrad_update<HIPContext>(
    int N,
    const float* w,
    const float* g,
    const float* h,
    float* nw,
    float* nh,
    float epsilon,
    float decay,
    const float* lr,
    HIPContext* context) {
  AdagradUpdate<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(N, w, g, h, nw, nh, epsilon, decay, lr);
}

template <typename SIndex, typename THalf>
__global__ void SparseAdagradKernel(
    const size_t N,
    const size_t grad_slice_sz,
    const float epsilon,
    THalf* param,
    THalf* param_mom,
    const SIndex* indices,
    const float* grad,
    const float* lr) {
  const float LR = lr[0];
  CUDA_1D_KERNEL_LOOP(i, N) {
    const size_t gradIdx = i;
    const SIndex index = indices[i / grad_slice_sz];
    const size_t paramIdx = index * grad_slice_sz + (i % grad_slice_sz);

    float mom_new = grad[gradIdx] * grad[gradIdx] + param_mom[paramIdx];
    param_mom[paramIdx] = mom_new;
    float param_new =
        LR * grad[gradIdx] / (sqrtf(mom_new) + epsilon) + param[paramIdx];
    param[paramIdx] = param_new;
  }
}

/**
 * Calculate RowwiseSparseAdagrad
 * M: gradients.dims[0]
 * N: gradients.size_from_dim(1)
 * grad: pointer to the gradients
 * param: pointer to weights
 * param_mom: pointer to the momentum
 * indices: keys
 */
template <typename SIndex>
__global__ void RowWiseSparseAdagradKernel(
    const int M,
    const int N,
    const float epsilon,
    float* param,
    float* param_mom,
    const SIndex* indices,
    const float* grad,
    const float* lr) {
  typedef hipcub::BlockReduce<float, CAFFE_CUDA_NUM_THREADS> BlockReduce;
  __shared__ BlockReduce::TempStorage temp_storage;
  int valid = min(N, CAFFE_CUDA_NUM_THREADS);
  // in case gridDim is smaller than M
  for (int i = blockIdx.x; i < M; i += gridDim.x) {
    const SIndex index = indices[i];
    float sum_squares = 0.0;
    __shared__ float row_sum_squares_avg;

    // in case N is bigger than block size which is 512 by default
    for (int j = threadIdx.x; j < N; j += blockDim.x) {
      const float x_ij = grad[i * N + j];
      sum_squares += x_ij * x_ij;
    }
    float reduce_result = BlockReduce(temp_storage).Sum(sum_squares, valid);
    if (threadIdx.x == 0) {
      row_sum_squares_avg = reduce_result / (float)N;
      param_mom[index] += row_sum_squares_avg;
    }
    __syncthreads();
    // update param
    float step = lr[0] / (sqrtf(param_mom[index]) + epsilon);
    for (int j = threadIdx.x; j < N; j += blockDim.x) {
      param[index * N + j] = param[index * N + j] + grad[i * N + j] * step;
    }
  }
}

template <typename T, class Context>
class CUDASparseAdagradOp final : public Operator<Context> {
 public:
  USE_OPERATOR_CONTEXT_FUNCTIONS;
  CUDASparseAdagradOp(const OperatorDef& operator_def, Workspace* ws)
      : Operator<Context>(operator_def, ws),
        epsilon_(this->template GetSingleArgument<float>("epsilon", 1e-5f)) {
    const T decay = this->template GetSingleArgument<T>("decay", 1.0f);
    CAFFE_ENFORCE_EQ(decay, 1.0, "Decay is not supported for SparseAdagradOp");
  }

  bool RunOnDevice() override {
    // Enforce shapes
    CAFFE_ENFORCE_EQ(Input(PARAM).size(), Input(MOMENT_1).size());
    CAFFE_ENFORCE_EQ(Input(LR).size(), 1);
    CAFFE_ENFORCE_EQ(
        Input(PARAM).size_from_dim(1),
        Input(GRAD).size_from_dim(Input(INDICES).ndim()));

    return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
        this, Input(INDICES));
  }

  template <typename IndexType>
  bool DoRunWithType() {
    auto n = Input(INDICES).size();
    if (n == 0) {
      return true;
    }
    return DispatchHelper<TensorTypes2<float, at::Half>, IndexType>::call(
        this, Input(PARAM));
  }

  template <typename IndexType, typename THalf>
  bool DoRunWithType2() {
    const auto* lr = Input(LR).template data<T>();
    const auto* indices = Input(INDICES).template data<IndexType>();
    const auto* gradIn = Input(GRAD).template data<T>();
    const auto* paramIn = Input(PARAM).template data<THalf>();
    const auto* momentIn = Input(MOMENT_1).template data<THalf>();
    auto* paramOut = Output(OUTPUT_PARAM)->template mutable_data<THalf>();
    auto* momentOut = Output(OUTPUT_MOMENT_1)->template mutable_data<THalf>();

    auto N = Input(GRAD).size();
    auto grad_slice_sz = Input(GRAD).size_from_dim(Input(INDICES).ndim());
    if (N == 0) {
      // empty grad, nothing to do here, not even launching the kernel
      return true;
    }
    SparseAdagradKernel<IndexType, THalf>
        <<<CAFFE_GET_BLOCKS(N),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context_.hip_stream()>>>(
            N,
            grad_slice_sz,
            epsilon_,
            Output(OUTPUT_PARAM)->template mutable_data<THalf>(),
            Output(OUTPUT_MOMENT_1)->template mutable_data<THalf>(),
            Input(INDICES).template data<IndexType>(),
            Input(GRAD).template data<float>(),
            Input(LR).template data<float>());
    return true;
  }

 protected:
  T epsilon_;
  INPUT_TAGS(PARAM, MOMENT_1, INDICES, GRAD, LR);
  OUTPUT_TAGS(OUTPUT_PARAM, OUTPUT_MOMENT_1);
};

template <>
template <typename SIndex>
bool RowWiseSparseAdagradOp<HIPContext>::DoRunWithType() {
  auto N = Input(GRAD).size();
  if (N == 0) {
    // empty grad, nothing to do here, not even launching the kernel
    return true;
  }
  // size of the 1st dimension of the input gradient
  auto GRAD_M = Input(GRAD).dim32(0);
  auto GRAD_N = N / GRAD_M;

  // Cases with GRAND_N < 128 can have more swarms if number of threads is lower
  int num_threads = CAFFE_CUDA_NUM_THREADS;
  if (GRAD_N < num_threads) {
    num_threads = GRAD_N;
  }

  // each thread block will handle multiple rows of the input and output
  RowWiseSparseAdagradKernel<<<
      std::min(GRAD_M, CAFFE_MAXIMUM_NUM_BLOCKS),
      num_threads,
      0,
      context_.hip_stream()>>>(
      GRAD_M,
      GRAD_N,
      epsilon_,
      Output(OUTPUT_PARAM)->template mutable_data<float>(),
      Output(OUTPUT_MOMENT_1)->template mutable_data<float>(),
      Input(INDICES).template data<SIndex>(),
      Input(GRAD).template data<float>(),
      Input(LR).template data<float>());
  return true;
}

template <typename T, typename TLengths, class Context>
class CUDASparseAdagradFusedWithSparseLengthsSumGradientOp final
    : public Operator<Context> {
 public:
  USE_OPERATOR_CONTEXT_FUNCTIONS;
  CUDASparseAdagradFusedWithSparseLengthsSumGradientOp(
      const OperatorDef& operator_def,
      Workspace* ws)
      : Operator<Context>(operator_def, ws),
        epsilon_(this->template GetSingleArgument<float>("epsilon", 1e-5f)) {
    const T decay = this->template GetSingleArgument<T>("decay", 1.0f);
    CAFFE_ENFORCE_EQ(decay, 1.0, "Decay is not supported for SparseAdagradOp");
  }

  bool RunOnDevice() override {
    // Enforce shapes
    CAFFE_ENFORCE_EQ(Input(PARAM).size(), Input(MOMENT_1).size());
    CAFFE_ENFORCE_EQ(Input(LR).size(), 1);
    CAFFE_ENFORCE_EQ(
        Input(PARAM).size_from_dim(1),
        Input(GRAD).size_from_dim(Input(INDICES).ndim()));

    return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
        this, Input(INDICES));
  }

  template <typename IndexType>
  bool DoRunWithType() {
    auto n = Input(INDICES).size();
    if (n == 0) {
      return true;
    }
    return DispatchHelper<TensorTypes2<float, at::Half>, IndexType>::call(
        this, Input(PARAM));
  }

  template <typename IndexType, typename THalf>
  bool DoRunWithType2() {
    auto& segmentGradsInput = Input(GRAD);
    auto& lengthsInput = Input(LENGTHS);
    auto& indicesInput = Input(INDICES);

    CAFFE_ENFORCE_EQ(1, lengthsInput.dim(), "LENGTHS must be a vector");
    CAFFE_ENFORCE_GT(Input(GRAD).dim(), 0);

    // Enforce:
    // input(embedding/momentum) == outputs(embedding/momentum)
    CAFFE_ENFORCE_EQ(
        Input(PARAM).numel(),
        Input(MOMENT_1).numel(),
        "Input Param size: ",
        Input(PARAM).numel(),
        " Input Moment size: ",
        Input(MOMENT_1).numel());

    const int len_length = lengthsInput.dim(0);
    CAFFE_ENFORCE(segmentGradsInput.dim() > 0);
    CAFFE_ENFORCE(len_length == segmentGradsInput.dim(0));

    int output_0dim = indicesInput.dim(0);

    if (len_length <= 0) {
      // return early to avoid invalid empty kernel
      return true;
    }

    inclusive_scan_length_buffer_.ResizeLike(lengthsInput);
    inclusive_scan_wrapper(
        lengthsInput.template data<int>(),
        len_length,
        &inclusive_scan_buffer_,
        &inclusive_scan_length_buffer_,
        &context_);

    // compute output size using length
    auto* prefix_sum_length_data =
        inclusive_scan_length_buffer_.template data<int>();

    const auto* lr = Input(LR).template data<T>();
    const auto* indices = Input(INDICES).template data<IndexType>();
    const T* grad = Input(GRAD).template data<T>();
    auto* paramOut = Output(OUTPUT_PARAM)->template mutable_data<THalf>();
    auto* momentOut = Output(OUTPUT_MOMENT_1)->template mutable_data<THalf>();

    int N = output_0dim;
    int post = segmentGradsInput.size_from_dim(1);

    auto maxThreads =
        GetDeviceProperty(CaffeCudaGetDevice()).maxThreadsPerBlock;

    if (post <= maxThreads) {
      int multiple = std::min(maxThreads / post, SEGREDUCE_MINBLOCKS);
      dim3 block(post, multiple);

      // calling cuda kernel with ExactBlock = true
      // T should be rename as TGRAD ?
      sparse_adagrad_fused_length_sum_gradient_kernel<IndexType, THalf, T, true>
          <<<len_length, block, 0, context_.hip_stream()>>>(
              prefix_sum_length_data,
              N,
              post,
              len_length,
              epsilon_,
              paramOut,
              momentOut,
              indices,
              grad,
              lr);
    } else {
      // calling cuda kernel with ExactBlock = false
      sparse_adagrad_fused_length_sum_gradient_kernel<
          IndexType,
          THalf,
          T,
          false><<<len_length, maxThreads, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          lr);
    }
    return true;
  }

 private:
  // menber field to manage memory
  Tensor inclusive_scan_buffer_{CUDA};
  Tensor inclusive_scan_length_buffer_{CUDA};

 protected:
  T epsilon_;
  INPUT_TAGS(PARAM, MOMENT_1, INDICES, GRAD, LR, LENGTHS);
  OUTPUT_TAGS(OUTPUT_PARAM, OUTPUT_MOMENT_1);
};

template <typename T, typename TLengths, class Context>
class CUDASparseAdagradFusedWithSparseLengthsWeightedSumGradientOp final
    : public Operator<Context> {
 public:
  USE_OPERATOR_CONTEXT_FUNCTIONS;
  CUDASparseAdagradFusedWithSparseLengthsWeightedSumGradientOp(
      const OperatorDef& operator_def,
      Workspace* ws)
      : Operator<Context>(operator_def, ws),
        epsilon_(this->template GetSingleArgument<float>("epsilon", 1e-5f)) {
    const T decay = this->template GetSingleArgument<T>("decay", 1.0f);
    CAFFE_ENFORCE_EQ(decay, 1.0, "Decay is not supported for SparseAdagradOp");
  }

  bool RunOnDevice() override {
    // Enforce shapes
    CAFFE_ENFORCE_EQ(Input(PARAM).size(), Input(MOMENT_1).size());
    CAFFE_ENFORCE_EQ(Input(LR).size(), 1);
    CAFFE_ENFORCE_EQ(
        Input(PARAM).size_from_dim(1),
        Input(GRAD).size_from_dim(Input(INDICES).ndim()));

    return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
        this, Input(INDICES));
  }

  template <typename IndexType>
  bool DoRunWithType() {
    auto n = Input(INDICES).size();
    if (n == 0) {
      // Allocate output to an empty tensor
      Output(AUX_GRAD, n, at::dtype<T>());
      return true;
    }
    return DispatchHelper<TensorTypes2<float, at::Half>, IndexType>::call(
        this, Input(PARAM));
  }

  template <typename IndexType, typename THalf>
  bool DoRunWithType2() {
    auto& segmentGradsInput = Input(GRAD);
    auto& lengthsInput = Input(LENGTHS);
    auto& indicesInput = Input(INDICES);
    auto& weightsInput = Input(AUX_PARAM);

    CAFFE_ENFORCE_EQ(1, lengthsInput.dim(), "LENGTHS must be a vector");
    CAFFE_ENFORCE_EQ(1, weightsInput.dim(), "WEIGHTS must be a vector");
    CAFFE_ENFORCE_GT(Input(GRAD).dim(), 0);

    // Enforce:
    // input(embedding/momentum) == outputs(embedding/momentum)
    CAFFE_ENFORCE_EQ(
        Input(PARAM).numel(),
        Input(MOMENT_1).numel(),
        "Input Param size: ",
        Input(PARAM).numel(),
        " Input Moment size: ",
        Input(MOMENT_1).numel());

    const int len_length = lengthsInput.dim(0);
    CAFFE_ENFORCE(segmentGradsInput.dim() > 0);
    CAFFE_ENFORCE(len_length == segmentGradsInput.dim(0));

    int output_0dim = indicesInput.dim(0);
    auto* weightGradsOutput =
        Output(AUX_GRAD, indicesInput.sizes(), at::dtype<T>());

    T* out_weight_grads = weightGradsOutput->template mutable_data<T>();

    if (len_length <= 0) {
      // return early to avoid invalid empty kernel
      return true;
    }

    inclusive_scan_length_buffer_.ResizeLike(lengthsInput);
    inclusive_scan_wrapper(
        lengthsInput.template data<int>(),
        len_length,
        &inclusive_scan_buffer_,
        &inclusive_scan_length_buffer_,
        &context_);

    // compute output size using length
    auto* prefix_sum_length_data =
        inclusive_scan_length_buffer_.template data<int>();

    const auto* lr = Input(LR).template data<T>();
    const auto* indices = Input(INDICES).template data<IndexType>();
    const T* grad = Input(GRAD).template data<T>();
    const T* weights = weightsInput.template data<T>();
    auto* paramOut = Output(OUTPUT_PARAM)->template mutable_data<THalf>();
    auto* momentOut = Output(OUTPUT_MOMENT_1)->template mutable_data<THalf>();

    int N = output_0dim;
    int post = segmentGradsInput.size_from_dim(1);

    auto maxThreads =
        GetDeviceProperty(CaffeCudaGetDevice()).maxThreadsPerBlock;

    if (post > 128) {
      sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          THalf,
          T,
          512><<<len_length, 512, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr);
    } else if (post > 64) {
      sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          THalf,
          T,
          128><<<len_length, 128, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr);
    } else if (post > 32) {
      sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          THalf,
          T,
          64><<<len_length, 64, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr);
    } else {
      sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          THalf,
          T,
          32><<<len_length, 32, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr);
    }
    return true;
  }

 private:
  // menber field to manage memory
  Tensor inclusive_scan_buffer_{CUDA};
  Tensor inclusive_scan_length_buffer_{CUDA};

 protected:
  T epsilon_;
  INPUT_TAGS(PARAM, MOMENT_1, AUX_PARAM, INDICES, GRAD, LR, LENGTHS);
  OUTPUT_TAGS(OUTPUT_PARAM, OUTPUT_MOMENT_1, AUX_GRAD);
};

template <typename T, typename TLengths, class Context>
class CUDARowWiseSparseAdagradFusedWithSparseLengthsSumGradientOp final
    : public Operator<Context> {
 public:
  USE_OPERATOR_CONTEXT_FUNCTIONS;
  CUDARowWiseSparseAdagradFusedWithSparseLengthsSumGradientOp(
      const OperatorDef& operator_def,
      Workspace* ws)
      : Operator<Context>(operator_def, ws),
        epsilon_(this->template GetSingleArgument<float>("epsilon", 1e-5f)) {
    const T decay = this->template GetSingleArgument<T>("decay", 1.0f);
    CAFFE_ENFORCE_EQ(decay, 1.0, "Decay is not supported for SparseAdagradOp");
  }

  bool RunOnDevice() override {
    // Enforce shapes
    CAFFE_ENFORCE_EQ(Input(LR).size(), 1);
    CAFFE_ENFORCE_EQ(
        Input(PARAM).size_from_dim(1),
        Input(GRAD).size_from_dim(Input(INDICES).ndim()));

    return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
        this, Input(INDICES));
  }

  template <typename IndexType>
  bool DoRunWithType() {
    auto n = Input(INDICES).size();
    if (n == 0) {
      return true;
    }
    return DispatchHelper<TensorTypes2<float, at::Half>, IndexType>::call(
        this, Input(PARAM));
  }

  template <typename IndexType, typename THalf>
  bool DoRunWithType2() {
    auto& segmentGradsInput = Input(GRAD);
    auto& lengthsInput = Input(LENGTHS);
    auto& indicesInput = Input(INDICES);

    CAFFE_ENFORCE_EQ(1, lengthsInput.dim(), "LENGTHS must be a vector");
    CAFFE_ENFORCE_GT(Input(GRAD).dim(), 0);

    // Enforce:
    // number of rows: input(embedding/momentum) == outputs(embedding/momentum)
    CAFFE_ENFORCE_EQ(
        Input(PARAM).dim(0),
        Input(MOMENT_1).dim(0),
        "Input Param number of rows: ",
        Input(PARAM).dim(0),
        " Input Moment size: ",
        Input(MOMENT_1).dim(0));

    const int len_length = lengthsInput.dim(0);
    CAFFE_ENFORCE(segmentGradsInput.dim() > 0);
    CAFFE_ENFORCE(len_length == segmentGradsInput.dim(0));

    int output_0dim = indicesInput.dim(0);

    if (len_length <= 0) {
      // return early to avoid invalid empty kernel
      return true;
    }

    inclusive_scan_length_buffer_.ResizeLike(lengthsInput);
    inclusive_scan_wrapper(
        lengthsInput.template data<int>(),
        len_length,
        &inclusive_scan_buffer_,
        &inclusive_scan_length_buffer_,
        &context_);

    // compute output size using length
    auto* prefix_sum_length_data =
        inclusive_scan_length_buffer_.template data<int>();

    const auto* lr = Input(LR).template data<T>();
    const auto* indices = Input(INDICES).template data<IndexType>();
    const T* grad = Input(GRAD).template data<T>();
    auto* paramOut = Output(OUTPUT_PARAM)->template mutable_data<THalf>();
    auto* momentOut = Output(OUTPUT_MOMENT_1)->template mutable_data<THalf>();

    int N = output_0dim;
    int post = segmentGradsInput.size_from_dim(1);

    auto maxThreads =
        GetDeviceProperty(CaffeCudaGetDevice()).maxThreadsPerBlock;

    rowwise_sparse_adagrad_fused_length_sum_gradient_kernel<IndexType, THalf, T>
        <<<len_length, std::min(maxThreads, post), 0, context_.hip_stream()>>>(
            prefix_sum_length_data,
            N,
            post,
            len_length,
            epsilon_,
            paramOut,
            momentOut,
            indices,
            grad,
            lr);

    return true;
  }

 private:
  // menber field to manage memory
  Tensor inclusive_scan_buffer_{CUDA};
  Tensor inclusive_scan_length_buffer_{CUDA};

 protected:
  T epsilon_;
  INPUT_TAGS(PARAM, MOMENT_1, INDICES, GRAD, LR, LENGTHS);
  OUTPUT_TAGS(OUTPUT_PARAM, OUTPUT_MOMENT_1);
};

template <typename T, typename TLengths, class Context>
class CUDARowWiseSparseAdagradFusedWithSparseLengthsWeightedSumGradientOp final
    : public Operator<Context> {
 public:
  USE_OPERATOR_CONTEXT_FUNCTIONS;
  CUDARowWiseSparseAdagradFusedWithSparseLengthsWeightedSumGradientOp(
      const OperatorDef& operator_def,
      Workspace* ws)
      : Operator<Context>(operator_def, ws),
        epsilon_(this->template GetSingleArgument<float>("epsilon", 1e-5f)) {
    const T decay = this->template GetSingleArgument<T>("decay", 1.0f);
    CAFFE_ENFORCE_EQ(decay, 1.0, "Decay is not supported for SparseAdagradOp");
  }

  bool RunOnDevice() override {
    // Enforce shapes
    CAFFE_ENFORCE_EQ(Input(LR).size(), 1);
    CAFFE_ENFORCE_EQ(
        Input(PARAM).size_from_dim(1),
        Input(GRAD).size_from_dim(Input(INDICES).ndim()));

    return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
        this, Input(INDICES));
  }

  template <typename IndexType>
  bool DoRunWithType() {
    auto n = Input(INDICES).size();
    if (n == 0) {
      Output(AUX_GRAD, n, at::dtype<T>());
      return true;
    }
    return DispatchHelper<TensorTypes2<float, at::Half>, IndexType>::call(
        this, Input(PARAM));
  }

  template <typename IndexType, typename THalf>
  bool DoRunWithType2() {
    auto& segmentGradsInput = Input(GRAD);
    auto& lengthsInput = Input(LENGTHS);
    auto& indicesInput = Input(INDICES);
    auto& weightsInput = Input(AUX_PARAM);

    CAFFE_ENFORCE_EQ(1, lengthsInput.dim(), "LENGTHS must be a vector");
    CAFFE_ENFORCE_EQ(1, weightsInput.dim(), "WEIGHTS must be a vector");
    CAFFE_ENFORCE_GT(Input(GRAD).dim(), 0);

    // Enforce:
    // number of rows: input(embedding/momentum) == outputs(embedding/momentum)
    CAFFE_ENFORCE_EQ(
        Input(PARAM).dim(0),
        Input(MOMENT_1).dim(0),
        "Input Param number of rows: ",
        Input(PARAM).dim(0),
        " Input Moment size: ",
        Input(MOMENT_1).dim(0));

    const int len_length = lengthsInput.dim(0);
    CAFFE_ENFORCE(segmentGradsInput.dim() > 0);
    CAFFE_ENFORCE(len_length == segmentGradsInput.dim(0));

    int output_0dim = indicesInput.dim(0);
    auto* weightGradsOutput =
        Output(AUX_GRAD, indicesInput.sizes(), at::dtype<T>());

    T* out_weight_grads = weightGradsOutput->template mutable_data<T>();

    if (len_length <= 0) {
      // return early to avoid invalid empty kernel
      return true;
    }

    inclusive_scan_length_buffer_.ResizeLike(lengthsInput);
    inclusive_scan_wrapper(
        lengthsInput.template data<int>(),
        len_length,
        &inclusive_scan_buffer_,
        &inclusive_scan_length_buffer_,
        &context_);

    // compute output size using length
    auto* prefix_sum_length_data =
        inclusive_scan_length_buffer_.template data<int>();

    const auto* lr = Input(LR).template data<T>();
    const auto* indices = Input(INDICES).template data<IndexType>();
    const T* grad = Input(GRAD).template data<T>();
    const T* weights = weightsInput.template data<T>();
    auto* paramOut = Output(OUTPUT_PARAM)->template mutable_data<THalf>();
    auto* momentOut = Output(OUTPUT_MOMENT_1)->template mutable_data<THalf>();

    int N = output_0dim;
    int post = segmentGradsInput.size_from_dim(1);

    auto maxThreads =
        GetDeviceProperty(CaffeCudaGetDevice()).maxThreadsPerBlock;

    if (post > 128) {
      rowwise_sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          THalf,
          T,
          512><<<len_length, 512, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr);
    } else if (post > 64) {
      rowwise_sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          THalf,
          T,
          128><<<len_length, 128, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr);
    } else if (post > 32) {
      rowwise_sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          THalf,
          T,
          64><<<len_length, 64, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr);
    } else {
      rowwise_sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          THalf,
          T,
          32><<<len_length, 32, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr);
    }

    return true;
  }

 private:
  // menber field to manage memory
  Tensor inclusive_scan_buffer_{CUDA};
  Tensor inclusive_scan_length_buffer_{CUDA};

 protected:
  T epsilon_;
  INPUT_TAGS(PARAM, MOMENT_1, AUX_PARAM, INDICES, GRAD, LR, LENGTHS);
  OUTPUT_TAGS(OUTPUT_PARAM, OUTPUT_MOMENT_1, AUX_GRAD);
};

REGISTER_HIP_OPERATOR(
    RowWiseSparseAdagradFusedWithSparseLengthsSumGradient,
    CUDARowWiseSparseAdagradFusedWithSparseLengthsSumGradientOp<
        float,
        int,
        HIPContext>);
REGISTER_HIP_OPERATOR(
    RowWiseSparseAdagradFusedWithSparseLengthsWeightedSumGradient,
    CUDARowWiseSparseAdagradFusedWithSparseLengthsWeightedSumGradientOp<
        float,
        int,
        HIPContext>);
REGISTER_HIP_OPERATOR(
    RowWiseSparseAdagradFusedWithSparseLengthsWeightedSumGradientApprox,
    CUDARowWiseSparseAdagradFusedWithSparseLengthsWeightedSumGradientOp<
        float,
        int,
        HIPContext>);

REGISTER_HIP_OPERATOR(
    SparseAdagradFusedWithSparseLengthsSumGradient,
    CUDASparseAdagradFusedWithSparseLengthsSumGradientOp<
        float,
        int,
        HIPContext>);
REGISTER_HIP_OPERATOR(
    SparseAdagradFusedWithSparseLengthsWeightedSumGradient,
    CUDASparseAdagradFusedWithSparseLengthsWeightedSumGradientOp<
        float,
        int,
        HIPContext>);
REGISTER_HIP_OPERATOR(
    SparseAdagradFusedWithSparseLengthsWeightedSumGradientApprox,
    CUDASparseAdagradFusedWithSparseLengthsWeightedSumGradientOp<
        float,
        int,
        HIPContext>);

REGISTER_HIP_OPERATOR(Adagrad, AdagradOp<HIPContext>);
REGISTER_HIP_OPERATOR(SparseAdagrad, CUDASparseAdagradOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    RowWiseSparseAdagrad,
    RowWiseSparseAdagradOp<HIPContext>);
} // namespace caffe2

#undef SEGREDUCE_MINBLOCKS
