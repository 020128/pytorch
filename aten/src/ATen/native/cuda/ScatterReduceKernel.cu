#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Dispatch.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>

#include "ScatterReduceReducer.cuh"

// TODO: how to define?
#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

namespace at { namespace native {

template <typename scalar_t, ReductionType REDUCE>
__global__ void
scatter_kernel(const scalar_t *src_data,
               const at::cuda::detail::TensorInfo<int64_t, int> index_info,
               scalar_t *out_data, int E, int K, int N, int numel) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  int b = thread_idx / (E * K);
  int k = thread_idx % K;

  if (thread_idx < numel) {
    int offset = at::cuda::detail::IndexToOffset<int64_t, int, -1>::get(
        thread_idx, index_info);
    int64_t idx = index_info.data[offset];

    Reducer<scalar_t, REDUCE>::atomic_write(out_data + b * N * K + idx * K + k,
                                            src_data[thread_idx]);
  }
}

template <typename scalar_t>
__global__ void
scatter_arg_kernel(const scalar_t *src_data,
                   const at::cuda::detail::TensorInfo<int64_t, int> index_info,
                   const scalar_t *out_data, int64_t *arg_out_data, int E,
                   int K, int N, int numel) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  int b = thread_idx / (E * K);
  int e = (thread_idx / K) % E;
  int k = thread_idx % K;

  if (thread_idx < numel) {
    int offset = at::cuda::detail::IndexToOffset<int64_t, int, -1>::get(
        thread_idx, index_info);
    int64_t idx = index_info.data[offset];

    if (src_data[thread_idx] == out_data[b * N * K + idx * K + k]) {
      arg_out_data[b * N * K + idx * K + k] = e;
    }
  }
}


Tensor scatter_reduce_two_cuda(const Tensor& self,
                               int64_t dim,
                               const Tensor& index,
                               const c10::string_view reduce,
                               const c10::optional<int64_t> output_size) {


  TORCH_CHECK(self.dim() == index.dim(),
      "Shape mismatch between `self` (got ", self.sizes(), ") and `index` (got ", index.sizes(), ")");
  for (int64_t i = 0; i < self.dim(); i++) {
    TORCH_CHECK(self.size(i) == index.size(i),
        "Shape mismatch between `self` (got ", self.sizes(), ") and `index` (got ", index.sizes(), ")");
  }

  auto self_cont = self.contiguous();
  auto index_cont = index.contiguous();

  auto sizes = self.sizes().vec();
  if (output_size.has_value())
    sizes[dim] = output_size.value();
  else if (index.numel() == 0)
    sizes[dim] = 0;
  else {
    auto d_size = index.max().data_ptr<int64_t>();
    auto h_size = (int64_t *)malloc(sizeof(int64_t));
    hipMemcpy(h_size, d_size, sizeof(int64_t), hipMemcpyDeviceToHost);
    sizes[dim] = 1 + *h_size;
  }

  Tensor out = at::empty(sizes, self.options());

  if (self.numel() == 0) {
    return out.zero_();
  }

  auto B = 1;
  for (auto i = 0; i < dim; i++)
    B *= self.size(i);
  auto E = self.size(dim);
  auto K = self.numel() / (B * E);
  auto N = out.size(dim);

  auto index_info = at::cuda::detail::getTensorInfo<int64_t, int>(index);
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(self.scalar_type(), "scatter_reduce", [&] {
    auto self_data = self_cont.data_ptr<scalar_t>();
    auto out_data = out.data_ptr<scalar_t>();

    AT_DISPATCH_REDUCTION_TYPES(reduce, [&] {
      out.fill_(Reducer<scalar_t, REDUCE>::init());

      scatter_kernel<scalar_t, REDUCE>
          <<<BLOCKS(self.numel()), THREADS, 0, stream>>>(
              self_data, index_info, out_data, E, K, N, self.numel());

      if (REDUCE == MIN || REDUCE == MAX)
        out.masked_fill_(out == Reducer<scalar_t, REDUCE>::init(), (scalar_t)0);
    });
  });

  return out;
}


}} // namespace at::native
